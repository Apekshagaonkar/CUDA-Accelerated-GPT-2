#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
using namespace std;

void isequal(float* a, float* b, int n){
    float maxval = -INFINITY;
    for(int i = 0;i<n;i++){
        maxval = fmaxf(maxval, fmaxf(a[i], b[i]));
    }
    float eps = 1e-5;
    for(int i = 0;i<n;i++){
        if(fabs(a[i] - b[i]) > eps * (maxval + 1)){
            cout << "Mismatch at index " << i << " CPU: " << a[i] << " GPU: " << b[i] << endl;
        }
    }
    cout << "Results match " << endl;
    for (int i = 0; i < 4; i++) {
        cout << a[i] << " " << b[i] << endl;
    }
}

void layernorm(float* out, float* x, float* w, float* b, int C){
    float mean = 0;
    float var = 0;
    for(int i = 0;i<C;i++){
        mean += x[i];
    }
    mean /= C;
    for(int i = 0;i<C;i++){
        float diff = x[i] - mean;
        var += diff * diff;
    }
    var /= C;
    float scale = 1.0 / sqrt(var + 1e-6);
    for(int i = 0;i<C;i++){
        out[i] = (x[i] - mean) * scale * w[i] + b[i];
    }
}

__global__
void layernorm_kernel(float* out, float* x, float* w, float* b, int C){
    int idx = threadIdx.x;
    float mean = 0;
    __shared__ float s_mean[1024];
    __shared__ float s_var[1024];
    s_mean[idx] = 0.0f;
    s_var[idx] = 0.0f;
    __syncthreads();

    for(int i = idx;i<C;i+=blockDim.x){
        s_mean[idx] += x[i];
    }
    __syncthreads();
    if(idx == 0){
        float m = 0;
        for(int i = 0;i<blockDim.x;i++){
            m += s_mean[i];
        }
        m /= C;
        s_mean[0] = m;
    }
    __syncthreads();
    mean = s_mean[0];

    for(int i = idx;i<C;i+=blockDim.x){
        float diff = x[i] - mean;
        s_var[idx] += diff * diff;
    }
    __syncthreads();
    if(idx == 0){
        float v = 0;
        for(int i = 0;i<blockDim.x;i++){
            v += s_var[i];
        }
        v /= C;
        s_var[0] = v;
    }
    __syncthreads();
    float var = s_var[0];
    float scale = 1.0 / sqrt(var + 1e-6);
    for(int i = idx;i<C;i+=blockDim.x){
        out[i] = (x[i] - mean) * scale * w[i] + b[i];
    }
}

void layernorm_gpu(float* out, float* x, float* w, float* b, int C){
    int numThreads = 256;
    int block = 1;
    layernorm_kernel<<<block,numThreads>>>(out,x,w,b,C);
}

int main(){

    int C = 768;
    float* x = (float*)malloc(C * sizeof(float));
    float* w = (float*)malloc(C * sizeof(float));
    float* b = (float*)malloc(C * sizeof(float));
    float* out = (float*)malloc(C * sizeof(float));

    float* x_gpu,*w_gpu,*b_gpu,*out_gpu;
    hipMalloc((void**)&x_gpu, C * sizeof(float));
    hipMalloc((void**)&w_gpu, C * sizeof(float));
    hipMalloc((void**)&b_gpu, C * sizeof(float));
    hipMalloc((void**)&out_gpu, C * sizeof(float));

    for (int i = 0; i < C; i++) {
        x[i] = i*0.92;
        w[i] = i;
        b[i] = i*0.5;
    }

    hipMemcpy(x_gpu, x, C * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(w_gpu, w, C * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, C * sizeof(float), hipMemcpyHostToDevice);

    auto start_cpu = std::chrono::high_resolution_clock::now();
    layernorm(out, x, w, b, C);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> cpu_duration = end_cpu - start_cpu;
    std::cout << "CPU Execution Time: " << cpu_duration.count() << " ms\n";

    // GPU Timing
    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start_gpu);
    layernorm_gpu(out_gpu, x_gpu, w_gpu, b_gpu, C);
    hipEventRecord(stop_gpu);

    hipEventSynchronize(stop_gpu);
    float gpu_duration = 0;
    hipEventElapsedTime(&gpu_duration, start_gpu, stop_gpu);
    std::cout << "GPU Execution Time: " << gpu_duration << " ms\n";

    float* check = (float*)malloc(C * sizeof(float));

    hipMemcpy(check, out_gpu, C * sizeof(float), hipMemcpyDeviceToHost);

    isequal(out, check, C);
    
    return 0;
}